#include "hip/hip_runtime.h"
#include <iostream>
#include <Eigen/Dense>
#include <Eigen/LU>

namespace cuda {

template < typename MatrixType, typename ResultType >
__device__ void inverse(MatrixType * matrix, ResultType * inverse)
{
    typedef typename MatrixType::Scalar Scalar;
    Eigen::internal::compute_inverse<MatrixType,ResultType>::run(*matrix,*inverse);
}

template < typename MatrixType >
__device__ void inverse_inplace(MatrixType * matrix)
{
    MatrixType inv;
    inverse(matrix,&inv);
    *matrix = inv;
}

}

template < typename MatrixType, typename ResultType >
__global__ void inverse_kernel(MatrixType * matrix, ResultType * inverse)
{
    cuda::inverse(matrix,inverse);
}

template < typename MatrixType >
__global__ void inverse_inplace_kernel(MatrixType * matrix)
{
    cuda::inverse_inplace(matrix);
}

int main(void)
{
    Eigen::Matrix3d mat = Eigen::Matrix3d::Random(4,4);

    // Host computaiton
    std::cout << mat.inverse() << '\n';

    // Device computaiton
    Eigen::Matrix3d inv;

    Eigen::Matrix3d * pmat;
    Eigen::Matrix3d * pinv;

    hipMalloc(&pmat, sizeof(Eigen::Matrix3d));
    hipMalloc(&pinv, sizeof(Eigen::Matrix3d));

    hipMemcpy(pmat, &mat, sizeof(Eigen::Matrix3d), hipMemcpyHostToDevice);

    inverse_kernel<<<1,1>>>(pmat,pinv);
    std::cerr << "CUDA exit status: " << hipGetErrorString(hipGetLastError()) << "\n";

    hipMemcpy(&inv, pinv, sizeof(Eigen::Matrix3d), hipMemcpyDeviceToHost);
    std::cout << inv << '\n';

    inverse_inplace_kernel<<<1,1>>>(pmat);
    std::cerr << "CUDA exit status: " << hipGetErrorString(hipGetLastError()) << "\n";

    hipMemcpy(&mat, pmat, sizeof(Eigen::Matrix3d), hipMemcpyDeviceToHost);
    std::cout << mat << '\n';
}
